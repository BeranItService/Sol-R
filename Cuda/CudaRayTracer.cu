#include "hip/hip_runtime.h"
﻿/* 
* Protein Visualizer
* Copyright (C) 2011-2012 Cyrille Favreau <cyrille_favreau@hotmail.com>
*
* This library is free software; you can redistribute it and/or
* modify it under the terms of the GNU Library General Public
* License as published by the Free Software Foundation; either
* version 2 of the License, or (at your option) any later version.
*
* This library is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
* Library General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

/*
* Author: Cyrille Favreau <cyrille_favreau@hotmail.com>
*
*/

// System
#include <iostream>

// Cuda
#include <hip/hip_runtime_api.h>
#if CUDART_VERSION>=5000
#include <hip/hip_runtime_api.h>
  #ifdef WIN32
    #include <hip/hip_vector_types.h>
  #else
    #include <cutil_math.h>
  #endif // WIN32
#else
#include <cutil_inline.h>
#include <cutil_math.h>
#endif


// Project
#include "CudaDataTypes.h"
#include "../Consts.h"

// Globals
#define M_PI 3.14159265358979323846
#define EPSILON 10.f

// Device arrays
Primitive*   d_primitives;
BoundingBox* d_boundingBoxes; 
int*         d_lamps;
Material*    d_materials;
char*        d_textures;
float*       d_randoms;
float4*      d_postProcessingBuffer;
char*        d_bitmap;
int*         d_primitivesXYIds;

// ________________________________________________________________________________
__device__ inline float vectorLength( const float4& vector )
{
	return sqrt( vector.x*vector.x + vector.y*vector.y + vector.z*vector.z );
}

// ________________________________________________________________________________
__device__ inline void normalizeVector( float4& v )
{
	v /= vectorLength( v );
}

// ________________________________________________________________________________
__device__ inline void saturateVector( float4& v )
{
	v.x = (v.x<0.f) ? 0.f : v.x;
	v.y = (v.y<0.f) ? 0.f : v.y; 
	v.z = (v.z<0.f) ? 0.f : v.z;
	v.w = (v.w<0.f) ? 0.f : v.w;

	v.x = (v.x>1.f) ? 1.f : v.x;
	v.y = (v.y>1.f) ? 1.f : v.y; 
	v.z = (v.z>1.f) ? 1.f : v.z;
	v.w = (v.w>1.f) ? 1.f : v.w;
}

// ________________________________________________________________________________
__device__ inline float dotProduct( const float4& v1, const float4& v2 )
{
	return ( v1.x*v2.x + v1.y*v2.y + v1.z*v2.z);
}

// ________________________________________________________________________________
__device__ inline float4 crossProduct( const float4& b, const float4& c )
{
	float4 a;
	a.x = b.y*c.z - b.z*c.y;
	a.y = b.z*c.x - b.x*c.z;
	a.z = b.x*c.y - b.y*c.x;
	return a;
}


/*
________________________________________________________________________________
incident  : le vecteur normal inverse a la direction d'incidence de la source 
lumineuse
normal    : la normale a l'interface orientee dans le materiau ou se propage le 
rayon incident
reflected : le vecteur normal reflechi
________________________________________________________________________________
*/
__device__ inline void vectorReflection( float4& r, const float4& i, const float4& n )
{
	r = i-2.f*dotProduct(i,n)*n;
}

__device__ float maxValue( const float& a, const float& b )
{
	return ( a>b ) ? a : b;
}

/*
________________________________________________________________________________
incident: le vecteur norm? inverse ? la direction d?incidence de la source 
lumineuse
n1      : index of refraction of original medium
n2      : index of refraction of new medium
________________________________________________________________________________
*/
__device__ inline void vectorRefraction( 
	float4&      refracted, 
	const float4 incident, 
	const float  n1, 
	const float4 normal, 
	const float  n2 )
{
	refracted = incident;
	if(n1!=n2 && n2!=0.f) 
	{
		float r = n1/n2;
		float cosI = dotProduct( incident, normal );
		float cosT2 = 1.f - r*r*(1.f - cosI*cosI);
		refracted = r*incident + (r*cosI-sqrt( fabs(cosT2) ))*normal;
	}
}

/*
________________________________________________________________________________
__v : Vector to rotate
__c : Center of rotations
__a : Angles
________________________________________________________________________________
*/
__device__ inline void vectorRotation( float4& vector, const float4 center, const float4 angles )
{ 
	float4 result = vector; 
	/* X axis */ 
	result.y = vector.y*cos(angles.x) - vector.z*sin(angles.x); 
	result.z = vector.y*sin(angles.x) + vector.z*cos(angles.x); 
	vector = result; 
	result = vector; 
	/* Y axis */ 
	result.z = vector.z*cos(angles.y) - vector.x*sin(angles.y); 
	result.x = vector.z*sin(angles.y) + vector.x*cos(angles.y); 
	vector = result; 
}

/*
________________________________________________________________________________

Compute ray attributes
________________________________________________________________________________
*/
__device__ inline void computeRayAttributes(Ray& ray)
{
	ray.inv_direction.x = 1.f/ray.direction.x;
	ray.inv_direction.y = 1.f/ray.direction.y;
	ray.inv_direction.z = 1.f/ray.direction.z;
	ray.signs.x = (ray.inv_direction.x < 0);
	ray.signs.y = (ray.inv_direction.y < 0);
	ray.signs.z = (ray.inv_direction.z < 0);
}

/*
________________________________________________________________________________

Convert float4 into OpenGL RGB color
________________________________________________________________________________
*/
__device__ void makeColor(
	const SceneInfo& sceneInfo,
	float4&   color,
	char*     bitmap,
	int       index)
{
   int mdc_index = index*gColorDepth; 
	color.x = (color.x>1.f) ? 1.f : color.x;
	color.y = (color.y>1.f) ? 1.f : color.y; 
	color.z = (color.z>1.f) ? 1.f : color.z;

   switch( sceneInfo.misc.x )
   {
      case otOpenGL: 
	   {
		   // OpenGL
		   bitmap[mdc_index  ] = (char)(color.x*255.f); // Red
      	bitmap[mdc_index+1] = (char)(color.y*255.f); // Green
		   bitmap[mdc_index+2] = (char)(color.z*255.f); // Blue
         break;
	   }
      case otDelphi: 
	   {
		   // Delphi
		   bitmap[mdc_index  ] = (char)(color.z*255.f); // Blue
      	bitmap[mdc_index+1] = (char)(color.y*255.f); // Green
		   bitmap[mdc_index+2] = (char)(color.x*255.f); // Red
         break;
	   }
      case otJPEG: 
	   {
         mdc_index = (sceneInfo.width.x*sceneInfo.height.x-index)*gColorDepth; 
		   // JPEG
		   bitmap[mdc_index+2] = (char)(color.z*255.f); // Blue
      	bitmap[mdc_index+1] = (char)(color.y*255.f); // Green
		   bitmap[mdc_index  ] = (char)(color.x*255.f); // Red
         break;
	   }
   }
}

__device__ void juliaSet( const SceneInfo& sceneInfo, const float x, const float y, float4& color )
{
   float W = (float)gTextureWidth;
   float H = (float)gTextureHeight;

   //pick some values for the constant c, this determines the shape of the Julia Set
   float cRe = -0.7f + 0.4f*sinf(sceneInfo.misc.y/1500.f);
   float cIm = 0.27015f + 0.4f*cosf(sceneInfo.misc.y/2000.f);

   //calculate the initial real and imaginary part of z, based on the pixel location and zoom and position values
   float newRe = 1.5f * (x - W / 2.f) / (0.5f * W);
   float newIm = (y - H / 2.f) / (0.5f * H);
   //i will represent the number of iterations
   int n;
   //start the iteration process
   float  maxIterations = 40.f+sceneInfo.pathTracingIteration.x;
   for(n = 0; n<maxIterations; n++)
   {
         //remember value of previous iteration
         float oldRe = newRe;
         float oldIm = newIm;
         //the actual iteration, the real and imaginary part are calculated
         newRe = oldRe * oldRe - oldIm * oldIm + cRe;
         newIm = 2.f * oldRe * oldIm + cIm;
         //if the point is outside the circle with radius 2: stop
         if((newRe * newRe + newIm * newIm) > 4.f) break;
   }
   //use color model conversion to get rainbow palette, make brightness black if maxIterations reached
   //color.x += newRe/4.f;
   //color.z += newIm/4.f;
   color.x = 1.f-color.x*(n/maxIterations);
   color.y = 1.f-color.y*(n/maxIterations);
   color.z = 1.f-color.z*(n/maxIterations);
}

__device__ void mandelbrotSet( const SceneInfo& sceneInfo, const float x, const float y, float4& color )
{
   float W = (float)gTextureWidth;
   float H = (float)gTextureHeight;

   float  MinRe		= -2.f;
   float  MaxRe		=	1.f;
   float  MinIm		= -1.2f;
   float  MaxIm		=	MinIm + (MaxRe - MinRe) * H/W;
   float  Re_factor	=	(MaxRe - MinRe) / (W - 1.f);
   double Im_factor	=	(MaxIm - MinIm) / (H - 1.f);
   float  maxIterations = 20.f+sceneInfo.pathTracingIteration.x;

   float c_im = MaxIm - y*Im_factor;
   float c_re = MinRe + x*Re_factor;
   float Z_re = c_re;
   float Z_im = c_im;
   bool isInside = true;
   unsigned n;
   for( n = 0; isInside && n < maxIterations; ++n ) 
   {
      float Z_re2 = Z_re*Z_re;
      float Z_im2 = Z_im*Z_im;
      if ( Z_re2+Z_im2>4.f ) 
      {
         isInside = false;
      }
      Z_im = 2.f*Z_re*Z_im+c_im;
      Z_re = Z_re2 - Z_im2+c_re;
   }

   //color.x += Z_re/64.f;
   //color.y += Z_im/64.f;
   color.x = 1.f-color.x*(n/maxIterations);
   color.y = 1.f-color.y*(n/maxIterations);
   color.z = 1.f-color.z*(n/maxIterations);
}

/*
________________________________________________________________________________

Sphere texture Mapping
________________________________________________________________________________
*/
__device__ float4 sphereUVMapping( 
	const Primitive& primitive,
	Material*        materials,
	char*            textures,
	const float4&    intersection)
{
	float4 result = materials[primitive.materialId.x].color;

	float4 d = primitive.p0-intersection;
	normalize(d);
	int u = primitive.size.x / primitive.materialInfo.x * (0.5f - atan2f(d.z, d.x) / 2*M_PI);
	int v = primitive.size.y / primitive.materialInfo.y * (0.5f - 2.f*(asinf(d.y) / 2*M_PI));

	u = u%gTextureWidth;
	v = v%gTextureHeight;
	if( u>=0 && u<gTextureWidth && v>=0 && v<gTextureHeight )
	{
		int index = gTextureOffset+(materials[primitive.materialId.x].textureInfo.y*gTextureWidth*gTextureHeight + v*gTextureWidth+u)*gTextureDepth;
		unsigned char r = textures[index  ];
		unsigned char g = textures[index+1];
		unsigned char b = textures[index+2];
		result.x = r/256.f;
		result.y = g/256.f;
		result.z = b/256.f;
	}
	return result; 
}

/*
________________________________________________________________________________

Cube texture mapping
________________________________________________________________________________
*/
__device__ float4 cubeMapping(
   const SceneInfo& sceneInfo,
	const Primitive& primitive, 
	Material*        materials,
	char*            textures,
	const float4&    intersection)
{
	float4 result = materials[primitive.materialId.x].color;

#ifdef USE_KINECT
	if( primitive.type.x == ptCamera )
	{
		int x = (intersection.x-primitive.p0.x+primitive.size.x)*primitive.materialInfo.x;
		int y = gKinectVideoHeight - (intersection.y-primitive.p0.y+primitive.size.y)*primitive.materialInfo.y;

		x = (x+gKinectVideoWidth)%gKinectVideoWidth;
		y = (y+gKinectVideoHeight)%gKinectVideoHeight;

		if( x>=0 && x<gKinectVideoWidth && y>=0 && y<gKinectVideoHeight ) 
		{
			int index = (y*gKinectVideoWidth+x)*gKinectVideo;
			unsigned char r = textures[index+2];
			unsigned char g = textures[index+1];
			unsigned char b = textures[index+0];
			result.x = r/256.f;
			result.y = g/256.f;
			result.z = b/256.f;
		}
	}
	else
#endif // USE_KINECT
	{
		int x = ((primitive.type.x == ptCheckboard) || (primitive.type.x == ptXZPlane) || (primitive.type.x == ptXYPlane))  ? 
			gTextureOffset+(intersection.x-primitive.p0.x+primitive.size.x)*primitive.materialInfo.x :
		gTextureOffset+(intersection.z-primitive.p0.z+primitive.size.z)*primitive.materialInfo.x;

		int y = ((primitive.type.x == ptCheckboard) || (primitive.type.x == ptXZPlane)) ? 
			gTextureOffset+(intersection.z+primitive.p0.z+primitive.size.z)*primitive.materialInfo.y :
		gTextureOffset+(intersection.y-primitive.p0.y+primitive.size.y)*primitive.materialInfo.y;

		x = x%gTextureWidth;
		y = y%gTextureHeight;

		if( x>=0 && x<gTextureWidth && y>=0 && y<gTextureHeight )
		{
         switch( materials[primitive.materialId.x].textureInfo.y )
         {
         case TEXTURE_MANDELBROT: mandelbrotSet( sceneInfo, x, y, result ); break;
         case TEXTURE_JULIA: juliaSet( sceneInfo, x, y, result ); break;
         default:
            {
			      int index = gTextureOffset+(materials[primitive.materialId.x].textureInfo.y*gTextureWidth*gTextureHeight + y*gTextureWidth+x)*gTextureDepth;
			      unsigned char r = textures[index];
			      unsigned char g = textures[index+1];
			      unsigned char b = textures[index+2];
			      result.x = r/256.f;
			      result.y = g/256.f;
			      result.z = b/256.f;
            }
            break;
         }
		}
	}
	return result;
}

__device__ bool wireFrameMapping( float x, float y, int width, const Primitive& primitive )
{
	int X = abs(x);
	int Y = abs(y);
	int A = primitive.materialInfo.x;
	int B = primitive.materialInfo.y;
	return ( X%A<=width ) || ( Y%B<=width );
}

#if 0
/*
________________________________________________________________________________

Magic Carpet texture mapping
________________________________________________________________________________
*/
__device__ float4 magicCarpetMapping( 
	Primitive primitive, 
	Material* materials,
	char*     textures,
	float4    intersection,
	int*      levels,
	float     timer)
{
	float4 result = materials[primitive.materialId.x].color;
	int x = gTextureOffset+(intersection.x-primitive.p0.x+primitive.size.x)*primitive.materialInfo.x*5.f;
	int y = gTextureOffset+(intersection.z+timer-primitive.p0.z+primitive.size.y)*primitive.materialInfo.y*50.f;

	x = x%gTextureWidth;
	y = y%gTextureHeight;

	if( x>=0 && x<gTextureWidth && y>=0 && y<gTextureHeight )
	{
		// Level management
		int tid_x = (intersection.x-primitive.p0.x+primitive.size.x      )/(primitive.size.x/2.5f);
		int tid_y = (intersection.z-primitive.p0.z+primitive.size.y+timer)/(primitive.size.y/25.f);
		int tid = tid_x+tid_y*5;
		tid = tid%5000;
		int index = (levels[tid]*gTextureWidth*gTextureHeight + y*gTextureWidth+x)*gTextureDepth;
		unsigned char r = textures[index];
		unsigned char g = textures[index+1];
		unsigned char b = textures[index+2];
		result.x = r/256.f;
		result.y = g/256.f;
		result.z = b/256.f;
	}
	return result;
}

/*
________________________________________________________________________________

Magic Cylinder texture mapping
________________________________________________________________________________
*/
__device__ float4 magicCylinderMapping( 
	Primitive primitive, 
	Material* materials,
	char*     textures,
	float4    intersection,
	int*      levels,
	float     timer)
{
	float4 result = materials[primitive.materialId.x].color;

	int x = gTextureOffset+(intersection.x-      primitive.p0.x+primitive.size.x)*primitive.materialInfo.x*5.f;
	int y = gTextureOffset+(intersection.z+timer-primitive.p0.z+primitive.size.y)*primitive.materialInfo.y*50.f;

	x = x%gTextureWidth;
	y = y%gTextureHeight;

	if( x>=0 && x<gTextureWidth && y>=0 && y<gTextureHeight )
	{
		int tid_x = (intersection.x-primitive.p0.x+primitive.size.x      )/(primitive.size.x/2.5f);
		int tid_y = (intersection.z-primitive.p0.z+primitive.size.y+timer)/(primitive.size.y/25.f);
		int tid = tid_x+tid_y*5;
		tid = tid%5000;
		int index = (levels[tid]*gTextureWidth*gTextureHeight + y*gTextureWidth+x)*gTextureDepth;
		unsigned char r = textures[index  ];
		unsigned char g = textures[index+1];
		unsigned char b = textures[index+2];
		result.x = r/256.f;
		result.y = g/256.f;
		result.z = b/256.f;
	}
	return result;
}
#endif // 0

/*
________________________________________________________________________________

Box intersection
________________________________________________________________________________
*/
__device__ inline bool boxIntersection( 
	const BoundingBox& box, 
	const Ray&     ray,
	const float    t0,
	const float    t1)
{
	float tmin, tmax, tymin, tymax, tzmin, tzmax;

	tmin = (box.parameters[ray.signs.x].x - ray.origin.x) * ray.inv_direction.x;
	tmax = (box.parameters[1-ray.signs.x].x - ray.origin.x) * ray.inv_direction.x;
	tymin = (box.parameters[ray.signs.y].y - ray.origin.y) * ray.inv_direction.y;
	tymax = (box.parameters[1-ray.signs.y].y - ray.origin.y) * ray.inv_direction.y;

	if ( (tmin > tymax) || (tymin > tmax) ) 
		return false;

	if (tymin > tmin) tmin = tymin;
	if (tymax < tmax) tmax = tymax;
	tzmin = (box.parameters[ray.signs.z].z - ray.origin.z) * ray.inv_direction.z;
	tzmax = (box.parameters[1-ray.signs.z].z - ray.origin.z) * ray.inv_direction.z;

	if ( (tmin > tzmax) || (tzmin > tmax) ) 
		return false;

	if (tzmin > tmin) tmin = tzmin;
	if (tzmax < tmax) tmax = tzmax;
	return ( (tmin < t1) && (tmax > t0) );
}

/*
________________________________________________________________________________

Ellipsoid intersection
________________________________________________________________________________
*/
__device__ inline bool ellipsoidIntersection(
	const SceneInfo& sceneInfo,
   const Primitive& ellipsoid,
	Material*  materials, 
   const Ray& ray, 
   float4& intersection,
   float4& normal,
	float& shadowIntensity,
   bool& back) 
{
	// Shadow intensity
	shadowIntensity = sceneInfo.shadowIntensity.x*(1.f-materials[ellipsoid.materialId.x].transparency.x);

   // solve the equation sphere-ray to find the intersections
	float4 O_C = ray.origin-ellipsoid.p0;
	float4 dir = ray.direction;
	normalizeVector( dir );

   float a = 
        ((dir.x*dir.x)/(ellipsoid.size.x*ellipsoid.size.x))
      + ((dir.y*dir.y)/(ellipsoid.size.y*ellipsoid.size.y))
      + ((dir.z*dir.z)/(ellipsoid.size.z*ellipsoid.size.z));
   float b = 
        ((2.f*O_C.x*dir.x)/(ellipsoid.size.x*ellipsoid.size.x))
      + ((2.f*O_C.y*dir.y)/(ellipsoid.size.y*ellipsoid.size.y))
      + ((2.f*O_C.z*dir.z)/(ellipsoid.size.z*ellipsoid.size.z));
   float c = 
        ((O_C.x*O_C.x)/(ellipsoid.size.x*ellipsoid.size.x))
      + ((O_C.y*O_C.y)/(ellipsoid.size.y*ellipsoid.size.y))
      + ((O_C.z*O_C.z)/(ellipsoid.size.z*ellipsoid.size.z))
      - 1.f;

   float d = ((b*b)-(4.f*a*c));
   if ( d<0.f || a==0.f || b==0.f || c==0.f ) 
   { 
      return false;
   }
   d = sqrt(d); 

   float t1 = (-b+d)/(2.f*a);
   float t2 = (-b-d)/(2.f*a);

	if( t1<=EPSILON && t2<=EPSILON ) return false; // both intersections are behind the ray origin
	back = (t1<=EPSILON || t2<=EPSILON); // If only one intersection (t>0) then we are inside the sphere and the intersection is at the back of the sphere

	float t=0.f;
	if( t1<=EPSILON ) 
		t = t2;
	else 
		if( t2<=EPSILON )
			t = t1;
		else
			t=(t1<t2) ? t1 : t2;

	if( t<EPSILON ) return false; // Too close to intersection
   intersection = ray.origin + t*dir;

   normal = intersection-ellipsoid.p0;
   normal.x = 2.f*normal.x/(ellipsoid.size.x*ellipsoid.size.x);
   normal.y = 2.f*normal.y/(ellipsoid.size.y*ellipsoid.size.y);
   normal.z = 2.f*normal.z/(ellipsoid.size.z*ellipsoid.size.z);

	normal.w = 0.f;
	normal *= (back) ? -1.f : 1.f;
	normalizeVector(normal);
   return true;
}


/*
________________________________________________________________________________

Sphere intersection
________________________________________________________________________________
*/
__device__ inline bool sphereIntersection(
	const SceneInfo& sceneInfo,
	const Primitive& sphere, 
	Material*  materials, 
	char*      textures, 
	const Ray& ray, 
	float4&    intersection,
	float4&    normal,
	float&     shadowIntensity,
	bool&      back
	) 
{
	// solve the equation sphere-ray to find the intersections
	float4 O_C = ray.origin-sphere.p0;
	float4 dir = ray.direction;
	normalizeVector( dir );

	float a = 2.f*dotProduct(dir,dir);
	float b = 2.f*dotProduct(O_C,dir);
	float c = dotProduct(O_C,O_C) - (sphere.size.x*sphere.size.x);
	float d = b*b-2.f*a*c;

	if( d<=0.f || a == 0.f) return false;
	float r = sqrt(d);
	float t1 = (-b-r)/a;
	float t2 = (-b+r)/a;

	if( t1<=EPSILON && t2<=EPSILON ) return false; // both intersections are behind the ray origin
	back = (t1<=EPSILON || t2<=EPSILON); // If only one intersection (t>0) then we are inside the sphere and the intersection is at the back of the sphere

	float t=0.f;
	if( t1<=EPSILON ) 
		t = t2;
	else 
		if( t2<=EPSILON )
			t = t1;
		else
			t=(t1<t2) ? t1 : t2;

	if( t<EPSILON ) return false; // Too close to intersection
	intersection = ray.origin+t*dir;

	// TO REMOVE - For Charts only
	//if( intersection.y < sphere.p0.y ) return false;

	// Shadow intensity
	shadowIntensity = sceneInfo.shadowIntensity.x*(1.f-materials[sphere.materialId.x].transparency.x);

	if( materials[sphere.materialId.x].textureInfo.x == 0) 
	{
		// Compute normal vector
		normal = intersection-sphere.p0;
	}
	else
	{
		// Procedural texture
		float4 newCenter;
      newCenter.x = sphere.p0.x + 0.008f*sphere.size.x*cos(sceneInfo.misc.y + intersection.x );
		newCenter.y = sphere.p0.y + 0.008f*sphere.size.y*sin(sceneInfo.misc.y + intersection.y );
		newCenter.z = sphere.p0.z + 0.008f*sphere.size.z*sin(cos(sceneInfo.misc.y + intersection.z ));
		normal  = intersection - newCenter;
	}
	normal.w = 0.f;
	normal *= (back) ? -1.f : 1.f;
	normalizeVector(normal);

#if EXTENDED_FEATURES
	// Power textures
	if (materials[sphere.materialId.x].textureInfo.y != TEXTURE_NONE && materials[sphere.materialId.x].transparency.x != 0 ) 
	{
		float4 color = sphereUVMapping(sphere, materials, textures, intersection, timer );
		return ((color.x+color.y+color.z) >= sceneInfo.transparentColor.x ); 
	}
#endif // 0

	return true;
}

/*
________________________________________________________________________________

Cylinder intersection
________________________________________________________________________________
*/
__device__ bool cylinderIntersection( 
	const SceneInfo& sceneInfo,
	const Primitive& cylinder,
	Material*  materials, 
	char*      textures,
	const Ray& ray,
	float4&    intersection,
	float4&    normal,
	float&     shadowIntensity,
	bool&      back) 
{
	back = false;
	float4 dir = ray.direction;
	/// normalizeVector(dir); // DO NOT NORMALIZE!!!
	float4 RC = ray.origin-cylinder.p0;
	float4 n = crossProduct(dir, cylinder.n1);

	float ln = vectorLength(n);

	// Parallel? (?)
	if((ln<EPSILON)&&(ln>-EPSILON))
		return false;

	normalizeVector(n);

	float d = fabs(dotProduct(RC,n));
	if (d>cylinder.p0.w) return false;

	float4 O = crossProduct(RC,cylinder.n1);
	float t = -dotProduct(O, n)/ln;
	O = crossProduct(n,cylinder.n1);
	normalizeVector(O);
	float s=fabs( sqrtf(cylinder.p0.w*cylinder.p0.w-d*d) / dotProduct( dir,O ) );

	float in=t-s;
	float out=t+s;

	if (in<-EPSILON)
	{
		if(out<-EPSILON)
			return false;
		else 
		{
			t=out;
			back = true;
		}
	}
	else
	{
		if(out<-EPSILON)
		{
			t=in;
		}
		else
		{
			if(in<out)
				t=in;
			else
			{
				t=out;
				back = true;
			}

			if( t<0.f ) return false;

			// Calculate intersection point
			intersection = ray.origin+t*dir;

			float4 HB1 = intersection-cylinder.p0;
			float4 HB2 = intersection-cylinder.p1;

			float scale1 = dotProduct(HB1,cylinder.n1);
			float scale2 = dotProduct(HB2,cylinder.n1);

			// Cylinder length
			if( scale1 < EPSILON || scale2 > EPSILON ) return false;

			if( materials[cylinder.materialId.x].textureInfo.x == 1) 
			{
				// Procedural texture
				float4 newCenter;
				newCenter.x = cylinder.p0.x + 0.01f*cylinder.size.x*cos(sceneInfo.misc.y/100.f+intersection.x);
				newCenter.y = cylinder.p0.y + 0.01f*cylinder.size.y*sin(sceneInfo.misc.y/100.f+intersection.y);
				newCenter.z = cylinder.p0.z + 0.01f*cylinder.size.z*sin(cos(sceneInfo.misc.y/100.f+intersection.z));
				HB1 = intersection - newCenter;
			}

			normal = HB1-cylinder.n1*scale1;
			normal.w = 0.f;

			normalizeVector( normal );

			// Shadow intensity
			shadowIntensity = sceneInfo.shadowIntensity.x*(1.f-materials[cylinder.materialId.x].transparency.x);
			return true;
		}
	}
   return false;
}

/*
________________________________________________________________________________

Checkboard intersection
________________________________________________________________________________
*/
__device__ bool planeIntersection( 
	const SceneInfo& sceneInfo,
	const Primitive& primitive,
	Material* materials,
	char*     textures,
	const Ray&      ray, 
	float4&   intersection,
	float4&   normal,
	float&    shadowIntensity,
	bool      reverse
	)
{ 
	bool collision = false;

	float reverted = reverse ? -1.f : 1.f;
	normal =  primitive.n0;
	switch( primitive.type.x ) 
	{
	case ptMagicCarpet:
	case ptCheckboard:
		{
			intersection.y = primitive.p0.y;
			float y = ray.origin.y-primitive.p0.y;
			if( reverted*ray.direction.y<0.f && reverted*ray.origin.y>reverted*primitive.p0.y) 
			{
				intersection.x = ray.origin.x+y*ray.direction.x/-ray.direction.y;
				intersection.z = ray.origin.z+y*ray.direction.z/-ray.direction.y;
				collision = 
					fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
					fabs(intersection.z - primitive.p0.z) < primitive.size.z;
			}
			break;
		}
	case ptXZPlane:
		{
			float y = ray.origin.y-primitive.p0.y;
			if( reverted*ray.direction.y<0.f && reverted*ray.origin.y>reverted*primitive.p0.y) 
			{
				intersection.x = ray.origin.x+y*ray.direction.x/-ray.direction.y;
				intersection.y = primitive.p0.y;
				intersection.z = ray.origin.z+y*ray.direction.z/-ray.direction.y;
				collision = 
					fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
					fabs(intersection.z - primitive.p0.z) < primitive.size.z;
				if( materials[primitive.materialId.x].textureInfo.z == 1 ) 
					collision &= wireFrameMapping(intersection.x, intersection.z, materials[primitive.materialId.x].textureInfo.w, primitive );
			}
			if( !collision && reverted*ray.direction.y>0.f && reverted*ray.origin.y<reverted*primitive.p0.y) 
			{
				normal = -normal;
				intersection.x = ray.origin.x+y*ray.direction.x/-ray.direction.y;
				intersection.y = primitive.p0.y;
				intersection.z = ray.origin.z+y*ray.direction.z/-ray.direction.y;
				collision = 
					fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
					fabs(intersection.z - primitive.p0.z) < primitive.size.z;
				if( materials[primitive.materialId.x].textureInfo.z == 1 ) 
					collision &= wireFrameMapping(intersection.x, intersection.z, materials[primitive.materialId.x].textureInfo.w, primitive );
			}
			break;
		}
	case ptYZPlane:
		{
			float x = ray.origin.x-primitive.p0.x;
			if( reverted*ray.direction.x<0.f && reverted*ray.origin.x>reverted*primitive.p0.x ) 
			{
				intersection.x = primitive.p0.x;
				intersection.y = ray.origin.y+x*ray.direction.y/-ray.direction.x;
				intersection.z = ray.origin.z+x*ray.direction.z/-ray.direction.x;
				collision = 
					fabs(intersection.y - primitive.p0.y) < primitive.size.y &&
					fabs(intersection.z - primitive.p0.z) < primitive.size.z;
				if( materials[primitive.materialId.x].textureInfo.z == 1 ) 
					collision &= wireFrameMapping(intersection.y, intersection.z, materials[primitive.materialId.x].textureInfo.w, primitive );
			}
			if( !collision && reverted*ray.direction.x>0.f && reverted*ray.origin.x<reverted*primitive.p0.x ) 
			{
				normal = -normal;
				intersection.x = primitive.p0.x;
				intersection.y = ray.origin.y+x*ray.direction.y/-ray.direction.x;
				intersection.z = ray.origin.z+x*ray.direction.z/-ray.direction.x;
				collision = 
					fabs(intersection.y - primitive.p0.y) < primitive.size.y &&
					fabs(intersection.z - primitive.p0.z) < primitive.size.z;
				if( materials[primitive.materialId.x].textureInfo.z == 1 ) 
					collision &= wireFrameMapping(intersection.y, intersection.z, materials[primitive.materialId.x].textureInfo.w, primitive );
			}
			break;
		}
	case ptXYPlane:
	case ptCamera:
		{
			float z = ray.origin.z-primitive.p0.z;
			if( reverted*ray.direction.z<0.f && reverted*ray.origin.z>reverted*primitive.p0.z) 
			{
				intersection.z = primitive.p0.z;
				intersection.x = ray.origin.x+z*ray.direction.x/-ray.direction.z;
				intersection.y = ray.origin.y+z*ray.direction.y/-ray.direction.z;
				collision = 
					fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
					fabs(intersection.y - primitive.p0.y) < primitive.size.y;
				if( materials[primitive.materialId.x].textureInfo.z == 1 ) 
					collision &= wireFrameMapping(intersection.x, intersection.y, materials[primitive.materialId.x].textureInfo.w, primitive );
			}
			if( !collision && reverted*ray.direction.z>0.f && reverted*ray.origin.z<reverted*primitive.p0.z )
			{
				normal = -normal;
				intersection.z = primitive.p0.z;
				intersection.x = ray.origin.x+z*ray.direction.x/-ray.direction.z;
				intersection.y = ray.origin.y+z*ray.direction.y/-ray.direction.z;
				collision = 
					fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
					fabs(intersection.y - primitive.p0.y) < primitive.size.y;
				if( materials[primitive.materialId.x].textureInfo.z == 1 ) 
					collision &= wireFrameMapping(intersection.x, intersection.y, materials[primitive.materialId.x].textureInfo.w, primitive );
			}
			break;
		}
	}

	if( collision ) 
	{
		// Shadow intensity
		shadowIntensity = sceneInfo.shadowIntensity.x*(1.f-materials[primitive.materialId.x].transparency.x);

		float4 color;
		color = materials[primitive.materialId.x].color;
		if( primitive.type.x == ptCamera || materials[primitive.materialId.x].textureInfo.y != TEXTURE_NONE )
		{
			color = cubeMapping(sceneInfo, primitive, materials, textures, intersection );
		}

		if( (color.x+color.y+color.z)/3.f >= sceneInfo.transparentColor.x ) 
		{
			collision = false;
		}
		else 
		{
			shadowIntensity = sceneInfo.shadowIntensity.x*
				(1.f-(color.x+color.y+color.z)/3.f*materials[primitive.materialId.x].transparency.x);
		}
	}
	return collision;
}

/*
________________________________________________________________________________

Triangle intersection
________________________________________________________________________________
*/
__device__ bool triangleIntersection( 
   const SceneInfo& sceneInfo,
	const Primitive& triangle, 
	Material* materials,
	const Ray&       ray,
	float4&          intersection,
	float4&          normal,
	float&           shadowIntensity
	) 
{
   // Reject rays using the barycentric coordinates of
   // the intersection point with respect to T.
   float4 E01 = triangle.p1 − triangle.p0;
   float4 E03 = triangle.p2 − triangle.p0;
   float4 P = crossProduct(ray.direction,E03);
   float det = dotProduct(E01,P);
   
   if (fabs(det) < EPSILON) return false;
   
   float4 T = ray.origin − triangle.p0;
   float a = dotProduct(T,P)/det;
   if (a < 0.f) return false;
   if (a > 1.f) return false;

   float4 Q = crossProduct(T,E01);
   float b = dotProduct(ray.direction,Q)/det;
   if (b < 0.f) return false;
   if (b > 1.f) return false;

   // Reject rays using the barycentric coordinates of
   // the intersection point with respect to T′.
   if ((a+b) > 1.f) 
   {
      float4 E23 = triangle.p0 − triangle.p1;
      float4 E21 = triangle.p1 − triangle.p1;
      float4 P_ = crossProduct(ray.direction,E21);
      float det_ = dotProduct(E23,P_);
      if(fabs(det_) < EPSILON) return false;
      float4 T_ = ray.origin − triangle.p2;
      float a_ = dotProduct(T_,P_)/det_;
      if (a_ < 0.f) return false;
      float4 Q_ = crossProduct(T_,E23);
      float b_ = dotProduct(ray.direction,Q_)/det_;
      if (b_ < 0.f) return false;
   }

   // Compute the ray parameter of the intersection
   // point.
   float t = dotProduct(E03,Q)/det;
   if (t < 0) return false;

   intersection = ray.origin + t*ray.direction;
   normal = triangle.n0;
   if( triangle.n0.w == 0.f )
   {
      float4 v0 = triangle.p0 - intersection;
      float4 v1 = triangle.p1 - intersection;
      float4 v2 = triangle.p2 - intersection;
      float a0 = 0.5f*vectorLength(crossProduct( v1,v2 ));
      float a1 = 0.5f*vectorLength(crossProduct( v0,v2 ));
      float a2 = 0.5f*vectorLength(crossProduct( v0,v1 ));
      normal = (triangle.n0*a0 + triangle.n1*a1 + triangle.n2*a2)/(a0+a1+a2);
   }
   normal *= (dotProduct(ray.direction,normal)>0.f) ? -1.f : 1.f;
	shadowIntensity = sceneInfo.shadowIntensity.x*(1.f-materials[triangle.materialId.x].transparency.x);
   return true;
}

/*
________________________________________________________________________________

Intersection Shader
________________________________________________________________________________
*/
__device__ float4 intersectionShader( 
	const SceneInfo& sceneInfo,
	const Primitive& primitive, 
	Material*    materials,
	char*        textures,
	const float4 intersection,
	const bool   back )
{
	float4 colorAtIntersection = materials[primitive.materialId.x].color;
	switch( primitive.type.x ) 
	{
	case ptCylinder:
		{
			if(materials[primitive.materialId.x].textureInfo.y != TEXTURE_NONE)
			{
				colorAtIntersection = sphereUVMapping(primitive, materials, textures, intersection );
			}
			break;
		}
	case ptEnvironment:
	case ptSphere:
   case ptEllipsoid:
		{
			if(materials[primitive.materialId.x].textureInfo.y != TEXTURE_NONE)
			{
				colorAtIntersection = sphereUVMapping(primitive, materials, textures, intersection );
			}
			break;
		}
	case ptCheckboard :
		{
			if( materials[primitive.materialId.x].textureInfo.y != TEXTURE_NONE ) 
			{
				colorAtIntersection = cubeMapping( sceneInfo, primitive, materials, textures, intersection );
			}
			else 
			{
				int x = sceneInfo.viewDistance.x + ((intersection.x - primitive.p0.x)/primitive.p0.w*primitive.materialInfo.x);
				int z = sceneInfo.viewDistance.x + ((intersection.z - primitive.p0.z)/primitive.p0.w*primitive.materialInfo.y);
				if(x%2==0) 
				{
					if (z%2==0) 
					{
						colorAtIntersection.x = 1.f-colorAtIntersection.x;
						colorAtIntersection.y = 1.f-colorAtIntersection.y;
						colorAtIntersection.z = 1.f-colorAtIntersection.z;
					}
				}
				else 
				{
					if (z%2!=0) 
					{
						colorAtIntersection.x = 1.f-colorAtIntersection.x;
						colorAtIntersection.y = 1.f-colorAtIntersection.y;
						colorAtIntersection.z = 1.f-colorAtIntersection.z;
					}
				}
			}
			break;
		}
	case ptXYPlane:
	case ptYZPlane:
	case ptXZPlane:
	case ptCamera:
		{
			if( materials[primitive.materialId.x].textureInfo.y != TEXTURE_NONE ) 
			{
				colorAtIntersection = cubeMapping( sceneInfo, primitive, materials, textures, intersection );
			}
			break;
		}
#if 0
	case ptTriangle:
		break;
	case ptMagicCarpet:
		{
			if( materials[primitive.materialId.x].textureInfo.y != TEXTURE_NONE ) 
			{
				colorAtIntersection = magicCarpetMapping( primitive, materials, textures, intersection, levels );
			}
			break;
		}
#endif // 0
	}
	return colorAtIntersection;
}

/*
________________________________________________________________________________

Shadows computation
We do not consider the object from which the ray is launched...
This object cannot shadow itself !

We now have to find the intersection between the considered object and the ray 
which origin is the considered 3D float4 and which direction is defined by the 
light source center.
.
. * Lamp                     Ray = Origin -> Light Source Center
.  \
.   \##
.   #### object
.    ##
.      \
.       \  Origin
.--------O-------
.
@return 1.f when pixel is in the shades

________________________________________________________________________________
*/
__device__ float processShadows(
	const SceneInfo& sceneInfo,
	BoundingBox*  boudingBoxes, const int& nbActiveBoxes,
	Primitive*    primitives,
	Material*     materials,
	char*         textures,
	const int&    nbPrimitives, 
	const float4& lampCenter, 
	const float4& origin, 
	const int&    objectId,
	const int&    iteration)
{
	float result = 0.f;
	int cptBoxes = 0;
	while( result<=1.f && cptBoxes < nbActiveBoxes )
	{
		Ray r;
		r.origin    = origin;
		r.direction = lampCenter-origin;
		//normalizeVector(r.direction); // TODO???
		computeRayAttributes( r );

		BoundingBox& box = boudingBoxes[cptBoxes];
		if( boxIntersection(box, r, 0.f, sceneInfo.viewDistance.x/iteration))
		{
			int cptPrimitives = 0;
			while( result<sceneInfo.shadowIntensity.x && cptPrimitives<box.nbPrimitives.x)
			{
				float4 intersection = {0.f,0.f,0.f,0.f};
				float4 normal       = {0.f,0.f,0.f,0.f};
				float  shadowIntensity = 0.f;

				if( (box.startIndex.x+cptPrimitives) != objectId )
				{
					Primitive& primitive = primitives[box.startIndex.x+cptPrimitives];

					bool hit = false;
					bool back;
					switch(primitive.type.x)
					{
					case ptSphere: 
						{
							hit = sphereIntersection  ( sceneInfo, primitive, materials, textures, r, intersection, normal, shadowIntensity, back ); 
							break;
						}
               case ptEllipsoid:
                  {
						   hit = ellipsoidIntersection( sceneInfo, primitive, materials, r, intersection, normal, shadowIntensity, back );
                     break;
                  }
					case ptCylinder:
						{
							hit = cylinderIntersection( sceneInfo, primitive, materials, textures, r, intersection, normal, shadowIntensity, back ); 
							break;
						}
					case ptTriangle:
						{
							hit = triangleIntersection( sceneInfo, primitive, materials, r, intersection, normal, shadowIntensity ); 
							break;
						}
					default:
						{
							hit = planeIntersection   ( sceneInfo, primitive, materials, textures, r, intersection, normal, shadowIntensity, false /*true*/ ); 
							break;
						}
					}

					if( hit )
					{
						float4 O_I = intersection-r.origin;
						float4 O_L = r.direction;
						float length = vectorLength(O_I);
						if( length>EPSILON && length<vectorLength(O_L) )
						{
							result += hit ? (shadowIntensity-materials[primitive.materialId.x].innerIllumination.x) : 0.f;
						}
					}
				}
				cptPrimitives++;
			}
		}
		cptBoxes++;
	}
	result = (result>1.f) ? 1.f : result;
	result = (result<0.f) ? 0.f : result;
	return result;
}

/*
________________________________________________________________________________

Primitive shader
________________________________________________________________________________
*/
__device__ float4 primitiveShader(
	const SceneInfo&   sceneInfo,
	const PostProcessingInfo&   postProcessingInfo,
	BoundingBox* boundingBoxes, int nbActiveBoxes,
	Primitive* primitives, const int& nbActivePrimitives,
	int* lamps, const int& nbActiveLamps,
	Material* materials, char* textures,
	float* randoms,
	const float4& origin,
	const float4& normal, 
	const int&    objectId, 
	const float4& intersection, 
	const int&    iteration,
	float4&       refractionFromColor,
	float&        shadowIntensity,
	float4&       totalBlinn)
{
	Primitive primitive = primitives[objectId];
	float4 color = materials[primitive.materialId.x].color;
	//color += materials[primitive.materialId.x].innerIllumination.x;
	//normalizeVector(color);

	float4 lampsColor = { 0.f, 0.f, 0.f, 0.f };

	// Lamp Impact
	float lambert      = 0.f;
	float totalLambert = (materials[primitive.materialId.x].innerIllumination.x != 0.f) ? 0.8f : sceneInfo.backgroundColor.w; // Ambient light
	shadowIntensity    = 0.f;

	if( materials[primitive.materialId.x].textureInfo.z == 1 )
		return color; //TODO? wireframe have constant color

	//if( sceneInfo.pathTracingIteration.x > 0 && materials[primitive.materialId.x].innerIllumination.x != 0.f ) 
	//   return color; 

	if( primitive.type.x == ptEnvironment )
	{
		// Final color
		color = intersectionShader( 
			sceneInfo, primitive, materials, textures, 
			intersection, false );
	}
	else 
	{
		color *= materials[primitive.materialId.x].innerIllumination.x;

		for( int cptLamps=0; cptLamps<nbActiveLamps; cptLamps++ ) 
		{
			if(lamps[cptLamps] != objectId)
			{
				float4 center;
				float4 size;
				switch( primitives[lamps[cptLamps]].type.x )
				{
				case ptCylinder:
					{
						center = (primitives[lamps[cptLamps]].p0 + primitives[lamps[cptLamps]].p1)/ 2.f;
						size.x = primitives[lamps[cptLamps]].size.y; 
						size.y = primitives[lamps[cptLamps]].size.y; 
						size.z = primitives[lamps[cptLamps]].size.y; 
						break;
					}
				default:
					{
						center = primitives[lamps[cptLamps]].p0; 
						size=primitives[lamps[cptLamps]].size; 
						break;
					}
				}

				if( sceneInfo.pathTracingIteration.x > 0 )
				{
					int t = 3*sceneInfo.pathTracingIteration.x + int(10.f*sceneInfo.misc.y)%100;
					// randomize lamp center
#if 0
					center.x += size.x*randoms[t  ]; //*sceneInfo.pathTracingIteration.x/float(sceneInfo.maxPathTracingIterations.x);
					center.y += size.y*randoms[t+1]; //*sceneInfo.pathTracingIteration.x/float(sceneInfo.maxPathTracingIterations.x);
					center.z += size.z*randoms[t+2]; //*sceneInfo.pathTracingIteration.x/float(sceneInfo.maxPathTracingIterations.x);
#else
					center.x += 10.f*size.x*randoms[t  ]*sceneInfo.pathTracingIteration.x/float(sceneInfo.maxPathTracingIterations.x);
					center.y += 10.f*size.y*randoms[t+1]*sceneInfo.pathTracingIteration.x/float(sceneInfo.maxPathTracingIterations.x);
					center.z += 10.f*size.z*randoms[t+2]*sceneInfo.pathTracingIteration.x/float(sceneInfo.maxPathTracingIterations.x);
#endif
				}

				if( sceneInfo.shadowsEnabled.x && materials[primitive.materialId.x].innerIllumination.x == 0.f ) 
				{
					shadowIntensity = processShadows(
						sceneInfo, boundingBoxes, nbActiveBoxes,
						primitives, materials, textures, 
						nbActivePrimitives, center, 
						intersection, lamps[cptLamps], iteration );
				}

				Material& material = materials[primitives[lamps[cptLamps]].materialId.x];
				float4 lightRay = center - intersection;
				normalizeVector(lightRay);

				// --------------------------------------------------------------------------------
				// Lambert
				// --------------------------------------------------------------------------------
				lambert = (postProcessingInfo.type.x==ppe_ambientOcclusion) ? 0.6f : dotProduct(normal,lightRay);
				lambert = (lambert<0.f) ? 0.f : lambert;
				lambert *= (materials[primitive.materialId.x].refraction.x == 0.f) ? material.innerIllumination.x : 1.f;
				lambert *= (1.f-shadowIntensity);
				totalLambert += lambert;

				// Lighted object, not in the shades
				lampsColor += lambert*material.color*material.innerIllumination.x;

				if( /*materials[primitive.materialId.x].innerIllumination.x == 0.f &&*/ shadowIntensity < sceneInfo.shadowIntensity.x )
				{
					// --------------------------------------------------------------------------------
					// Blinn - Phong
					// --------------------------------------------------------------------------------
					float4 viewRay = intersection - origin;
					normalizeVector(viewRay);

					float4 blinnDir = lightRay - viewRay;
					float temp = sqrt(dotProduct(blinnDir,blinnDir));
					if (temp != 0.f ) 
					{
						// Specular reflection
						blinnDir = (1.f / temp) * blinnDir;

						float blinnTerm = dotProduct(blinnDir,normal);
						blinnTerm = ( blinnTerm < 0.f) ? 0.f : blinnTerm;

						blinnTerm = materials[primitive.materialId.x].specular.x * pow(blinnTerm,materials[primitive.materialId.x].specular.y); //*materials[primitive.materialId.x].specular.w
						totalBlinn += material.color * material.innerIllumination.x * blinnTerm;
					}
				}
			}
		}
		// Final color
		float4 intersectionColor = 
			intersectionShader( sceneInfo, primitive, materials, textures,
			intersection, false );

		color += /*totalLambert**/ intersectionColor*lampsColor;
		saturateVector(color);

		refractionFromColor = intersectionColor; // Refraction depending on color;
		saturateVector( totalBlinn );
	}
	return color;
}

/*
________________________________________________________________________________

Intersections with primitives
________________________________________________________________________________
*/
__device__ bool intersectionWithPrimitives(
	const SceneInfo& sceneInfo,
	BoundingBox* boundingBoxes, const int& nbActiveBoxes,
	Primitive* primitives, const int& nbActivePrimitives,
	Material* materials, char* textures,
	const Ray& ray, 
	const int& iteration,
	int&    closestPrimitive, 
	float4& closestIntersection,
	float4& closestNormal,
	float4& colorBox,
	bool&   back,
   const int currentMaterialId)
{
	bool intersections = false; 
	float minDistance  = sceneInfo.viewDistance.x;

	Ray r;
	r.origin    = ray.origin;
	r.direction = ray.direction-ray.origin;
	computeRayAttributes( r );

	float4 intersection = {0.f,0.f,0.f,0.f};
	float4 normal       = {0.f,0.f,0.f,0.f};
	bool i = false;
	float shadowIntensity = 0.f;

	for( int cptBoxes = 0; cptBoxes<nbActiveBoxes; ++cptBoxes )
	{
		BoundingBox& box = boundingBoxes[cptBoxes];
		if( boxIntersection(box, r, 0.f, sceneInfo.viewDistance.x/iteration) )
		{
			// Intersection with Box
			if( sceneInfo.renderBoxes.x != 0 ) 
         {
            colorBox += materials[cptBoxes%NB_MAX_MATERIALS].color / 10.f;
         }

			// Intersection with primitive within boxes
			for( int cptPrimitives = 0; cptPrimitives<box.nbPrimitives.x; ++cptPrimitives )
			{ 
				Primitive& primitive = primitives[box.startIndex.x+cptPrimitives];
            Material& material = materials[primitive.materialId.x];
            if( material.fastTransparency.x==0 || (material.fastTransparency.x==1 && currentMaterialId != primitive.materialId.x)) // !!!! TEST SHALL BE REMOVED TO INCREASE TRANSPARENCY QUALITY !!!
            {
				   i = false;
				   switch( primitive.type.x )
				   {
				   case ptEnvironment :
               case ptSphere:
                  {
						   i = sphereIntersection  ( sceneInfo, primitive, materials, textures, r, intersection, normal, shadowIntensity, back ); 
						   break;
					   }
				   case ptCylinder: 
					   {
						   i = cylinderIntersection( sceneInfo, primitive, materials, textures, r, intersection, normal, shadowIntensity, back ); 
						   break;
					   }
               case ptEllipsoid:
                  {
						   i = ellipsoidIntersection( sceneInfo, primitive, materials, r, intersection, normal, shadowIntensity, back );
                     break;
                  }
               case ptTriangle:
                  {
						   back = false;
						   i = triangleIntersection( sceneInfo, primitive, materials, r, intersection, normal, shadowIntensity ); 
                     break;
                  }
				   default: 
					   {
                     back = false;
						   i = planeIntersection   ( sceneInfo, primitive, materials, textures, r, intersection, normal, shadowIntensity, false); 
						   break;
					   }
				   }

				   float distance = vectorLength( intersection - r.origin ); // <- Pb ici!!
				   if( i && distance>EPSILON && distance<minDistance ) 
				   {
					   // Only keep intersection with the closest object
					   minDistance         = distance;
					   closestPrimitive    = box.startIndex.x+cptPrimitives;
					   closestIntersection = intersection;
					   closestNormal       = normal;
					   intersections       = true;
				   }
            }
			}
		}
	}
	return intersections;
}

/*
________________________________________________________________________________

Calculate the reflected vector                   

^ Normal to object surface (N)  
Reflection (O_R)  |                              
\ |  Eye (O_E)                    
\| /                             
----------------O--------------- Object surface 
closestIntersection                      

============================================================================== 
colours                                                                                    
------------------------------------------------------------------------------ 
We now have to know the colour of this intersection                                        
Color_from_object will compute the amount of light received by the
intersection float4 and  will also compute the shadows. 
The resulted color is stored in result.                     
The first parameter is the closest object to the intersection (following 
the ray). It can  be considered as a light source if its inner light rate 
is > 0.                            
________________________________________________________________________________
*/
__device__ float4 launchRay( 
	BoundingBox* boundingBoxes, const int& nbActiveBoxes,
	Primitive* primitives, const int& nbActivePrimitives,
	int* lamps, const int& nbActiveLamps,
	Material*  materials, char* textures,
	float*           randoms,
	const Ray&       ray, 
	const SceneInfo& sceneInfo,
	const PostProcessingInfo& postProcessingInfo,
	float4&          intersection,
	float&           depthOfField,
	int&             primitiveXYId)
{
	float4 intersectionColor   = {0.f,0.f,0.f,0.f};

	float4 closestIntersection = {0.f,0.f,0.f,0.f};
	float4 firstIntersection   = {0.f,0.f,0.f,0.f};
	float4 normal              = {0.f,0.f,0.f,0.f};
	int    closestPrimitive  = 0;
	bool   carryon           = true;
	Ray    rayOrigin         = ray;
	float  initialRefraction = 1.f;
	int    iteration         = 0;
	float  previousWeight    = 1.f;
	primitiveXYId = -1;

	float4 recursiveColor = { 0.f, 0.f, 0.f, 0.f };
	float4 recursiveBlinn = { 0.f, 0.f, 0.f, 0.f };

	// Variable declarations
	float  shadowIntensity = 0.f;
	float4 refractionFromColor;
	float4 reflectedTarget;
	float4 colorBox = { 0.f, 0.f, 0.f, 0.f };
	bool   back = false;

   int currentMaterialId=-2;

	while( iteration<(sceneInfo.nbRayIterations.x+sceneInfo.pathTracingIteration.x) && carryon ) 
	{
		// If no intersection with lamps detected. Now compute intersection with Primitives
		if( carryon ) 
		{
			carryon = intersectionWithPrimitives(
				sceneInfo,
				boundingBoxes, nbActiveBoxes,
				primitives, nbActivePrimitives,
				materials, textures,
				rayOrigin,
				iteration,  
				closestPrimitive, closestIntersection, 
				normal, colorBox, back, currentMaterialId);
		}

		if( carryon ) 
		{
         currentMaterialId = primitives[closestPrimitive].materialId.x;

			if ( iteration==0 )
			{
				intersectionColor.x = 0.f;
				intersectionColor.y = 0.f;
				intersectionColor.z = 0.f;
				intersectionColor.w = 0.f;

				firstIntersection = closestIntersection;
				primitiveXYId = closestPrimitive;
			}

         float recursiveColorRatio = 1.f;
			float4 rBlinn = {0.f,0.f,0.f,0.f};

			if( shadowIntensity <= 0.9f ) // No reflection/refraction if in shades
			{
				// ----------
				// Refraction
				// ----------

				if( materials[primitives[closestPrimitive].materialId.x].transparency.x != 0.f ) 
				{
					// Replace the normal using the intersection color
					// r,g,b become x,y,z... What the fuck!!
					if( materials[primitives[closestPrimitive].materialId.x].textureInfo.y != TEXTURE_NONE) 
					{
						normal *= (recursiveColor-0.5f);
					}

					// Back of the object? If so, reset refraction to 1.f (air)
					float refraction = back ? 1.f : materials[primitives[closestPrimitive].materialId.x].refraction.x;

					// Actual refraction
					float4 O_E = rayOrigin.origin - closestIntersection;
					normalizeVector(O_E);
					vectorRefraction( rayOrigin.direction, O_E, refraction, normal, initialRefraction );
					reflectedTarget = closestIntersection - rayOrigin.direction;

               recursiveColorRatio = previousWeight*(1.f-materials[primitives[closestPrimitive].materialId.x].transparency.x);
					previousWeight = previousWeight*materials[primitives[closestPrimitive].materialId.x].transparency.x;

               // Prepare next ray
					initialRefraction = refraction;
				}
				else
				{
					// ----------
					// Reflection
					// ----------
					if( materials[primitives[closestPrimitive].materialId.x].reflection.x != 0.f ) 
					{
						float4 O_E = rayOrigin.origin - closestIntersection;
						vectorReflection( rayOrigin.direction, O_E, normal );

						reflectedTarget = closestIntersection - rayOrigin.direction;
						recursiveColorRatio = previousWeight*(1.f-materials[primitives[closestPrimitive].materialId.x].reflection.x);
						previousWeight = previousWeight*materials[primitives[closestPrimitive].materialId.x].reflection.x;
					}
					else 
					{
						recursiveColorRatio *= previousWeight;
						carryon = false;
					}         
				}
			}
			else 
			{
				recursiveColorRatio *= previousWeight;
				carryon = false;
			}

			// Get object color
			recursiveColor = primitiveShader( 
				sceneInfo, postProcessingInfo,
				boundingBoxes, nbActiveBoxes,
			   primitives, nbActivePrimitives, lamps, nbActiveLamps, materials, textures, 
			   randoms,
			   rayOrigin.origin, normal, closestPrimitive, closestIntersection, 
			   iteration, refractionFromColor, shadowIntensity, rBlinn );

			// Contribute to final color
         intersectionColor += recursiveColor*recursiveColorRatio;
 			recursiveBlinn += rBlinn;
			intersectionColor -= colorBox;

         rayOrigin.origin    = closestIntersection + reflectedTarget*0.00001f; 
			rayOrigin.direction = reflectedTarget;


			// Noise management
			if( sceneInfo.pathTracingIteration.x != 0 && materials[primitives[closestPrimitive].materialId.x].color.w != 0.f)
			{
				// Randomize view
				int rindex = 3.f*sceneInfo.misc.y + sceneInfo.pathTracingIteration.x;
				rindex = rindex%(sceneInfo.width.x*sceneInfo.height.x);
				rayOrigin.direction.x += randoms[rindex  ]*materials[primitives[closestPrimitive].materialId.x].color.w;
				rayOrigin.direction.y += randoms[rindex+1]*materials[primitives[closestPrimitive].materialId.x].color.w;
				rayOrigin.direction.z += randoms[rindex+2]*materials[primitives[closestPrimitive].materialId.x].color.w;
			}
		}
		else
		{
			intersectionColor += previousWeight*sceneInfo.backgroundColor;
		}
		iteration++; 
	}

	intersectionColor += recursiveBlinn;

	saturateVector( intersectionColor );
	intersection = closestIntersection;

	float len = vectorLength(firstIntersection - ray.origin);
   float halfDistance = sceneInfo.viewDistance.x*0.75f;
   if( sceneInfo.misc.z==1 && len>halfDistance)
   {
	   // --------------------------------------------------
	   // Attenation effect (Fog)
	   // --------------------------------------------------
	   len = 1.f-((len-halfDistance)/(sceneInfo.viewDistance.x-halfDistance));
	   len = (len>0.f) ? len : 0.f; 
	   len = (len<1.f) ? len : 1.f; 
	   intersectionColor.x *= len;
	   intersectionColor.y *= len;
	   intersectionColor.z *= len;
	   
      intersectionColor.x += sceneInfo.backgroundColor.x*(1.f-len);
	   intersectionColor.y += sceneInfo.backgroundColor.y*(1.f-len);
	   intersectionColor.z += sceneInfo.backgroundColor.z*(1.f-len);
   }

	// Depth of field
   depthOfField = (len-depthOfField)/sceneInfo.viewDistance.x;
	return intersectionColor;
}


/*
________________________________________________________________________________

Standard renderer
________________________________________________________________________________
*/
__global__ void k_standardRenderer(
	BoundingBox* BoundingBoxes, int nbActiveBoxes,
	Primitive* primitives, int nbActivePrimitives,
	int* lamps, int nbActiveLamps,
	Material*    materials,
	char*        textures,
	float*       randoms,
	float4       origin,
	float4       direction,
	float4       angles,
	SceneInfo    sceneInfo,
	PostProcessingInfo postProcessingInfo,
	float4*      postProcessingBuffer,
	int*         primitiveXYIds)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;

	Ray ray;
	ray.origin = origin;
	ray.direction = direction;

	float4 rotationCenter = {0.f,0.f,0.f,0.f};

	if( sceneInfo.pathTracingIteration.x == 0 )
   {
		postProcessingBuffer[index].x = 0.f;
		postProcessingBuffer[index].y = 0.f;
		postProcessingBuffer[index].z = 0.f;
		postProcessingBuffer[index].w = 0.f;
   }
   else
	{
		// Randomize view
		int rindex = index + sceneInfo.pathTracingIteration.x;
		rindex = rindex%(sceneInfo.width.x*sceneInfo.height.x);
		ray.direction.x += randoms[rindex  ]*postProcessingBuffer[index].w*postProcessingInfo.param2.x*float(sceneInfo.pathTracingIteration.x)/float(sceneInfo.maxPathTracingIterations.x);
		ray.direction.y += randoms[rindex+1]*postProcessingBuffer[index].w*postProcessingInfo.param2.x*float(sceneInfo.pathTracingIteration.x)/float(sceneInfo.maxPathTracingIterations.x);
		ray.direction.z += randoms[rindex+2]*postProcessingBuffer[index].w*postProcessingInfo.param2.x*float(sceneInfo.pathTracingIteration.x)/float(sceneInfo.maxPathTracingIterations.x);
	}

	float dof = postProcessingInfo.param1.x;
	float4 intersection;


   if( sceneInfo.misc.w == 1 ) // Isometric 3D
   {
      //ray.origin.z = (ray.origin.z<0.f ) ? ray.origin.z : 0.f;
      ray.direction.x = /* ray.direction.x - (ray.origin.z*0.01f)* */ ray.origin.z*0.001f*(float)(x - (sceneInfo.width.x/2));
	   ray.direction.y = /* ray.direction.y - (ray.origin.z*0.01f)* */ -ray.origin.z*0.001f*(float)(y - (sceneInfo.height.x/2));
      //ray.direction.z = 3000.f;
	   ray.origin.x = ray.direction.x;
	   ray.origin.y = ray.direction.y;
      //ray.origin.z = -5000.f;
   }
   else
   {
      float ratio=(float)sceneInfo.width.x/(float)sceneInfo.height.x;
      float2 step;
      step.x=ratio*6400.f/(float)sceneInfo.width.x;
      step.y=6400.f/(float)sceneInfo.height.x;
      ray.direction.x = ray.direction.x - step.x*(float)(x - (sceneInfo.width.x/2));
      ray.direction.y = ray.direction.y + step.y*(float)(y - (sceneInfo.height.x/2));
   }


	vectorRotation( ray.origin, rotationCenter, angles );
	vectorRotation( ray.direction, rotationCenter, angles );

   __shared__ BoundingBox shBoundingBoxes[128];
   if( threadIdx.x==0 && threadIdx.y==0)
   {
      memcpy( shBoundingBoxes, BoundingBoxes, sizeof(BoundingBox)*nbActiveBoxes);
   }
   __syncthreads();

	float4 color = launchRay(
		shBoundingBoxes, nbActiveBoxes,
		primitives, nbActivePrimitives,
		lamps, nbActiveLamps,
		materials, textures, 
		randoms,
		ray, 
		sceneInfo, postProcessingInfo,
		intersection,
		dof,
		primitiveXYIds[index]);

	if( sceneInfo.pathTracingIteration.x == 0 )
	{
		postProcessingBuffer[index].w = dof;
	}
   postProcessingBuffer[index].x += color.x;
   postProcessingBuffer[index].y += color.y;
   postProcessingBuffer[index].z += color.z;
}

/*
________________________________________________________________________________

Anaglyph Renderer
________________________________________________________________________________
*/
__global__ void k_anaglyphRenderer(
	BoundingBox* BoundingBoxes, int nbActiveBoxes,
	Primitive* primitives, int nbActivePrimitives,
	int* lamps, int nbActiveLamps,
	Material*    materials,
	char*        textures,
	float*       randoms,
	float4       origin,
	float4       direction,
	float4       angles,
	SceneInfo    sceneInfo,
	PostProcessingInfo postProcessingInfo,
	float4*      postProcessingBuffer,
	int*         primitiveXYIds)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;

	float4 rotationCenter = {0.f,0.f,0.f,0.f};

	if( sceneInfo.pathTracingIteration.x == 0 )
	{
		postProcessingBuffer[index].x = 0.f;
		postProcessingBuffer[index].y = 0.f;
		postProcessingBuffer[index].z = 0.f;
		postProcessingBuffer[index].w = 0.f;
	}

	float dof = postProcessingInfo.param1.x;
	float4 intersection;
	Ray eyeRay;

	// Left eye
	eyeRay.origin.x = origin.x + sceneInfo.width3DVision.x;
	eyeRay.origin.y = origin.y;
	eyeRay.origin.z = origin.z;

	eyeRay.direction.x = direction.x - 8.f*(float)(x - (sceneInfo.width.x/2));
	eyeRay.direction.y = direction.y + 8.f*(float)(y - (sceneInfo.height.x/2));
	eyeRay.direction.z = direction.z;

	vectorRotation( eyeRay.origin, rotationCenter, angles );
	vectorRotation( eyeRay.direction, rotationCenter, angles );

	float4 colorLeft = launchRay(
		BoundingBoxes, nbActiveBoxes,
		primitives, nbActivePrimitives,
		lamps, nbActiveLamps,
		materials, textures, 
		randoms,
		eyeRay, 
		sceneInfo, postProcessingInfo,
		intersection,
		dof,
		primitiveXYIds[index]);

	// Right eye
	eyeRay.origin.x = origin.x - sceneInfo.width3DVision.x;
	eyeRay.origin.y = origin.y;
	eyeRay.origin.z = origin.z;

	eyeRay.direction.x = direction.x - 8.f*(float)(x - (sceneInfo.width.x/2));
	eyeRay.direction.y = direction.y + 8.f*(float)(y - (sceneInfo.height.x/2));
	eyeRay.direction.z = direction.z;

	vectorRotation( eyeRay.origin, rotationCenter, angles );
	vectorRotation( eyeRay.direction, rotationCenter, angles );
	float4 colorRight = launchRay(
		BoundingBoxes, nbActiveBoxes,
		primitives, nbActivePrimitives,
		lamps, nbActiveLamps,
		materials, textures, 
		randoms,
		eyeRay, 
		sceneInfo, postProcessingInfo,
		intersection,
		dof,
		primitiveXYIds[index]);

	float r1 = colorLeft.x*0.299f + colorLeft.y*0.587f + colorLeft.z*0.114f;
	float b1 = 0.f;
	float g1 = 0.f;

	float r2 = 0.f;
	float g2 = colorRight.y;
	float b2 = colorRight.z;

	postProcessingBuffer[index].x += r1+r2;
	postProcessingBuffer[index].y += g1+g2;
	postProcessingBuffer[index].z += b1+b2;
	if( sceneInfo.pathTracingIteration.x == 0 ) postProcessingBuffer[index].w = dof;
}

/*
________________________________________________________________________________

3D Vision Renderer
________________________________________________________________________________
*/
__global__ void k_3DVisionRenderer(
	BoundingBox* BoundingBoxes, int nbActiveBoxes,
	Primitive*   primitives,    int nbActivePrimitives,
	int* lamps, int nbActiveLamps,
	Material*    materials,
	char*        textures,
	float*       randoms,
	float4       origin,
	float4       direction,
	float4       angles,
	SceneInfo    sceneInfo,
	PostProcessingInfo postProcessingInfo,
	float4*      postProcessingBuffer,
	int*         primitiveXYIds)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;

	float4 rotationCenter = {0.f,0.f,0.f,0.f};

	if( sceneInfo.pathTracingIteration.x == 0 )
	{
		postProcessingBuffer[index].x = 0.f;
		postProcessingBuffer[index].y = 0.f;
		postProcessingBuffer[index].z = 0.f;
		postProcessingBuffer[index].w = 0.f;
	}

	float dof = postProcessingInfo.param1.x;
	float4 intersection;
	int halfWidth  = sceneInfo.width.x/2;

	Ray eyeRay;
	if( x<halfWidth ) 
	{
		// Left eye
		eyeRay.origin.x = origin.x + sceneInfo.width3DVision.x;
		eyeRay.origin.y = origin.y;
		eyeRay.origin.z = origin.z;

		eyeRay.direction.x = direction.x - 8.f*(float)(x - (sceneInfo.width.x/2) + halfWidth/2 );
		eyeRay.direction.y = direction.y + 8.f*(float)(y - (sceneInfo.height.x/2));
		eyeRay.direction.z = direction.z;
	}
	else
	{
		// Right eye
		eyeRay.origin.x = origin.x - sceneInfo.width3DVision.x;
		eyeRay.origin.y = origin.y;
		eyeRay.origin.z = origin.z;

		eyeRay.direction.x = direction.x - 8.f*(float)(x - (sceneInfo.width.x/2) - halfWidth/2);
		eyeRay.direction.y = direction.y + 8.f*(float)(y - (sceneInfo.height.x/2));
		eyeRay.direction.z = direction.z;
	}

	vectorRotation( eyeRay.origin, rotationCenter, angles );
	vectorRotation( eyeRay.direction, rotationCenter, angles );

	float4 color = launchRay(
		BoundingBoxes, nbActiveBoxes,
		primitives, nbActivePrimitives,
		lamps, nbActiveLamps,
		materials, textures, 
		randoms,
		eyeRay, 
		sceneInfo, postProcessingInfo,
		intersection,
		dof,
		primitiveXYIds[index]);

	postProcessingBuffer[index].x += color.x;
	postProcessingBuffer[index].y += color.y;
	postProcessingBuffer[index].z += color.z;
	if( sceneInfo.pathTracingIteration.x == 0 ) postProcessingBuffer[index].w = dof;
}


/*
________________________________________________________________________________

Post Processing Effect: Depth of field
________________________________________________________________________________
*/
__global__ void k_depthOfField(
	SceneInfo        sceneInfo,
	PostProcessingInfo PostProcessingInfo,
	float4*          postProcessingBuffer,
	float*           randoms,
	char*            bitmap) 
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;
	float  depth = PostProcessingInfo.param2.x*postProcessingBuffer[index].w;
	int    wh = sceneInfo.width.x*sceneInfo.height.x;

	float4 localColor;
	localColor.x = 0.f;
	localColor.y = 0.f;
	localColor.z = 0.f;

	for( int i=0; i<PostProcessingInfo.param3.x; ++i )
	{
		int ix = i%wh;
		int iy = (i+sceneInfo.width.x)%wh;
		int xx = x+depth*randoms[ix]*0.1f;
		int yy = y+depth*randoms[iy]*0.1f;
		if( xx>=0 && xx<sceneInfo.width.x && yy>=0 && yy<sceneInfo.height.x )
		{
			int localIndex = yy*sceneInfo.width.x+xx;
			if( localIndex>=0 && localIndex<wh )
			{
				localColor += postProcessingBuffer[localIndex];
			}
		}
		else
		{
			localColor += postProcessingBuffer[index];
		}
	}
	localColor /= PostProcessingInfo.param3.x;
	localColor /= (sceneInfo.pathTracingIteration.x+1);
	localColor.w = 1.f;

	makeColor( sceneInfo, localColor, bitmap, index ); 
}

/*
________________________________________________________________________________

Post Processing Effect: Ambiant Occlusion
________________________________________________________________________________
*/
__global__ void k_ambiantOcclusion(
	SceneInfo        sceneInfo,
	PostProcessingInfo postProcessingInfo,
	float4*          postProcessingBuffer,
	float*           randoms,
	char*            bitmap) 
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;
	float occ = 0.f;
	float4 localColor = postProcessingBuffer[index];
	float  depth = localColor.w;

   const int step = 16;
	for( int X=-step; X<step; X+=2 )
	{
		for( int Y=-step; Y<step; Y+=2 )
		{
			int xx = x+X;
			int yy = y+Y;
			if( xx>=0 && xx<sceneInfo.width.x && yy>=0 && yy<sceneInfo.height.x )
			{
				int localIndex = yy*sceneInfo.width.x+xx;
				if( postProcessingBuffer[localIndex].w>=depth)
				{
					occ += 1.f;
				}
			}
			else
				occ += 1.f;
		}
	}
	//occ /= float((2*step)*(2*step));
	occ /= float(step*step);
	occ += 0.3f; // Ambient light
	localColor.x *= occ;
	localColor.y *= occ;
	localColor.z *= occ;
	localColor /= (sceneInfo.pathTracingIteration.x+1);
	saturateVector( localColor );
	localColor.w = 1.f;

	makeColor( sceneInfo, localColor, bitmap, index ); 
}

/*
________________________________________________________________________________

Post Processing Effect: Cartoon
________________________________________________________________________________
*/
__global__ void k_cartoon(
	SceneInfo        sceneInfo,
	PostProcessingInfo PostProcessingInfo,
	float4*          postProcessingBuffer,
	float*           randoms,
	char*            bitmap) 
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;
	float4 localColor = postProcessingBuffer[index];

	int r = localColor.x*255/PostProcessingInfo.param3.x;
	int g = localColor.y*255/PostProcessingInfo.param3.x;
	int b = localColor.z*255/PostProcessingInfo.param3.x;

	localColor.x = float(r*PostProcessingInfo.param3.x/255.f);
	localColor.y = float(g*PostProcessingInfo.param3.x/255.f);
	localColor.z = float(b*PostProcessingInfo.param3.x/255.f);
	localColor /= (sceneInfo.pathTracingIteration.x+1);

	localColor.w = 1.f;
	makeColor( sceneInfo, localColor, bitmap, index ); 
}

/*
________________________________________________________________________________

Post Processing Effect: Ambiant Occlusion
________________________________________________________________________________
*/
__global__ void k_antiAliasing(
	SceneInfo        sceneInfo,
	PostProcessingInfo PostProcessingInfo,
	float4*          postProcessingBuffer,
	float*           randoms,
	char*            bitmap) 
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;
	float4 localColor = {0.f,0.f,0.f,0.f};

	for( int X=-1; X<=1; X+=2 )
	{
		for( int Y=-1; Y<=1; Y+=2 )
		{
			int xx = x+X;
			int yy = y+Y;
			if( xx>=0 && xx<sceneInfo.width.x && yy>=0 && yy<sceneInfo.height.x )
			{
				int localIndex = yy*sceneInfo.width.x+xx;
				localColor += 0.2f*postProcessingBuffer[localIndex];
			}
		}
	}
	localColor /= 4.f;
	localColor += postProcessingBuffer[index];
	localColor /= (sceneInfo.pathTracingIteration.x+1);
	saturateVector( localColor );
	localColor.w = 1.f;

	makeColor( sceneInfo, localColor, bitmap, index ); 
}

/*
________________________________________________________________________________

Post Processing Effect: Default
________________________________________________________________________________
*/
__global__ void k_default(
	SceneInfo        sceneInfo,
	PostProcessingInfo PostProcessingInfo,
	float4*          postProcessingBuffer,
	char*            bitmap) 
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int index = y*sceneInfo.width.x+x;

   float4 localColor = postProcessingBuffer[index]/(float)(sceneInfo.pathTracingIteration.x+1.f);

	makeColor( sceneInfo, localColor, bitmap, index ); 
}

/*
________________________________________________________________________________

GPU initialization
________________________________________________________________________________
*/
extern "C" void initialize_scene( 
	int width, int height, int nbPrimitives, int nbLamps, int nbMaterials, int nbTextures )
{
	// Scene resources
	checkCudaErrors(hipMalloc( (void**)&d_boundingBoxes,      NB_MAX_BOXES*sizeof(BoundingBox)));
	checkCudaErrors(hipMalloc( (void**)&d_primitives,         NB_MAX_PRIMITIVES*sizeof(Primitive)));
	checkCudaErrors(hipMalloc( (void**)&d_lamps,              NB_MAX_LAMPS*sizeof(int)));
	checkCudaErrors(hipMalloc( (void**)&d_materials,          NB_MAX_MATERIALS*sizeof(Material)));
	checkCudaErrors(hipMalloc( (void**)&d_textures,           NB_MAX_TEXTURES*gTextureDepth*gTextureWidth*gTextureHeight + gTextureOffset));
	checkCudaErrors(hipMalloc( (void**)&d_randoms,            width*height*sizeof(float)));

	// Rendering canvas
	checkCudaErrors(hipMalloc( (void**)&d_postProcessingBuffer,  width*height*sizeof(float4)));
	checkCudaErrors(hipMalloc( (void**)&d_bitmap,                width*height*gColorDepth*sizeof(char)));
	checkCudaErrors(hipMalloc( (void**)&d_primitivesXYIds,       width*height*gColorDepth*sizeof(int)));

#if 0
	std::cout <<"GPU: SceneInfo         : " << sizeof(SceneInfo) << std::endl;
	std::cout <<"GPU: Ray               : " << sizeof(Ray) << std::endl;
	std::cout <<"GPU: PrimitiveType     : " << sizeof(PrimitiveType) << std::endl;
	std::cout <<"GPU: Material          : " << sizeof(Material) << std::endl;
	std::cout <<"GPU: BoundingBox       : " << sizeof(BoundingBox) << std::endl;
	std::cout <<"GPU: Primitive         : " << sizeof(Primitive) << std::endl;
	std::cout <<"GPU: PostProcessingType: " << sizeof(PostProcessingType) << std::endl;
	std::cout <<"GPU: PostProcessingInfo: " << sizeof(PostProcessingInfo) << std::endl;
	std::cout <<"Textures " << NB_MAX_TEXTURES << std::endl;
#endif // 0
}

/*
________________________________________________________________________________

GPU finalization
________________________________________________________________________________
*/
extern "C" void finalize_scene()
{
	checkCudaErrors(hipFree( d_boundingBoxes ));
	checkCudaErrors(hipFree( d_primitives ));
	checkCudaErrors(hipFree( d_lamps ));
	checkCudaErrors(hipFree( d_materials ));
	checkCudaErrors(hipFree( d_textures ));
	checkCudaErrors(hipFree( d_randoms ));
	checkCudaErrors(hipFree( d_postProcessingBuffer ));
	checkCudaErrors(hipFree( d_bitmap ));
	checkCudaErrors(hipFree( d_primitivesXYIds ));
}

/*
________________________________________________________________________________

CPU -> GPU data transfers
________________________________________________________________________________
*/
extern "C" void h2d_scene( 
	BoundingBox* boundingBoxes, int nbActiveBoxes,
	Primitive*  primitives, int nbPrimitives,
	int* lamps, int nbLamps )
{
	checkCudaErrors(hipMemcpy( d_boundingBoxes,      boundingBoxes,      nbActiveBoxes*sizeof(BoundingBox), hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_primitives,         primitives,         nbPrimitives*sizeof(Primitive),    hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_lamps,              lamps,              nbLamps*sizeof(int),               hipMemcpyHostToDevice ));
}

extern "C" void h2d_materials( 
	Material*  materials, int nbActiveMaterials,
	char*      textures , int nbActiveTextures,
	float*     randoms,   int nbRandoms)
{
	checkCudaErrors(hipMemcpy( d_materials, materials, nbActiveMaterials*sizeof(Material), hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_textures,  textures,  gTextureOffset+nbActiveTextures*sizeof(char)*gTextureSize,  hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_randoms,   randoms,   nbRandoms*sizeof(float), hipMemcpyHostToDevice ));
}

#ifdef USE_KINECT
extern "C" void h2d_kinect( 
	char* kinectVideo, char* kinectDepth )
{
	checkCudaErrors(hipMemcpy( d_textures, kinectVideo, gKinectVideoSize*sizeof(char), hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_textures+gKinectVideoSize, kinectDepth, gKinectDepthSize*sizeof(char), hipMemcpyHostToDevice ));
}
#endif // USE_KINECT

/*
________________________________________________________________________________

GPU -> CPU data transfers
________________________________________________________________________________
*/
extern "C" void d2h_bitmap( char* bitmap, int* primitivesXYIds, const SceneInfo sceneInfo )
{
	checkCudaErrors(hipMemcpy( bitmap, d_bitmap, sceneInfo.width.x*sceneInfo.height.x*gColorDepth*sizeof(char), hipMemcpyDeviceToHost ));
	checkCudaErrors(hipMemcpy( primitivesXYIds, d_primitivesXYIds, sceneInfo.width.x*sceneInfo.height.x*sizeof(int), hipMemcpyDeviceToHost ));
}

/*
________________________________________________________________________________

Kernel launcher
________________________________________________________________________________
*/
extern "C" void cudaRender(
	int4 blockSize, int sharedMemSize,
	SceneInfo sceneInfo,
	int4 objects,
	PostProcessingInfo postProcessingInfo,
	float4 origin, 
	float4 direction, 
	float4 angles)
{
	int2 size;
	size.x = static_cast<int>(sceneInfo.width.x);
	size.y = static_cast<int>(sceneInfo.height.x);

	dim3 grid((size.x+blockSize.x-1)/blockSize.x,(size.y+blockSize.y-1)/blockSize.y,1);
	dim3 blocks( blockSize.x,blockSize.y,blockSize.z );
   sharedMemSize = objects.x*sizeof(BoundingBox);

	switch( sceneInfo.supportFor3DVision.x ) 
	{
	case vtAnaglyph:
		{
			k_anaglyphRenderer<<<grid,blocks,sharedMemSize>>>(
				d_boundingBoxes, objects.x, d_primitives, objects.y,  d_lamps, objects.z, d_materials, d_textures, 
				d_randoms, origin, direction, angles, sceneInfo, 
				postProcessingInfo, d_postProcessingBuffer, d_primitivesXYIds);
			break;
		}
	case vt3DVision:
		{
			k_3DVisionRenderer<<<grid,blocks,sharedMemSize>>>(
				d_boundingBoxes, objects.x, d_primitives, objects.y,  d_lamps, objects.z, d_materials, d_textures, 
				d_randoms, origin, direction, angles, sceneInfo, 
				postProcessingInfo, d_postProcessingBuffer, d_primitivesXYIds);
			break;
		}
	default:
		{
			k_standardRenderer<<<grid,blocks,sharedMemSize>>>(
				d_boundingBoxes, objects.x, d_primitives, objects.y,  d_lamps, objects.z, d_materials, d_textures, 
				d_randoms, origin, direction, angles, sceneInfo,
				postProcessingInfo, d_postProcessingBuffer, d_primitivesXYIds);
			break;
		}
	}

	hipDeviceSynchronize();
	hipError_t status = hipGetLastError();
	if(status != hipSuccess) 
	{
		std::cout << "ERROR: (" << status << ") " << hipGetErrorString(status) << std::endl;
		std::cout << "INFO: Size(" << size.x << ", " << size.y << ") " << std::endl;
		std::cout << "INFO: Grid(" << grid.x << ", " << grid.y << ", " << grid.z <<") " << std::endl;
		std::cout << "nbActiveBoxes :" << objects.x << std::endl;
		std::cout << "nbActivePrimitives :" << objects.y << std::endl;
		std::cout << "nbActiveLamps :" << objects.z << std::endl;
	}

	switch( postProcessingInfo.type.x )
	{
	case ppe_depthOfField:
		k_depthOfField<<<grid,blocks>>>(
			sceneInfo, 
			postProcessingInfo, 
			d_postProcessingBuffer,
			d_randoms, 
			d_bitmap );
		break;
	case ppe_ambientOcclusion:
		k_ambiantOcclusion<<<grid,blocks>>>(
			sceneInfo, 
			postProcessingInfo, 
			d_postProcessingBuffer,
			d_randoms, 
			d_bitmap );
		break;
	case ppe_cartoon:
		k_cartoon<<<grid,blocks>>>(
			sceneInfo, 
			postProcessingInfo, 
			d_postProcessingBuffer,
			d_randoms, 
			d_bitmap );
		break;
	case ppe_antiAliasing:
		k_antiAliasing<<<grid,blocks>>>(
			sceneInfo, 
			postProcessingInfo, 
			d_postProcessingBuffer,
			d_randoms, 
			d_bitmap );
		break;
	default:
		k_default<<<grid,blocks>>>(
			sceneInfo, 
			postProcessingInfo, 
			d_postProcessingBuffer,
			d_bitmap );
		break;
	}
}
